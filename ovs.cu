#include "hip/hip_runtime.h"
#define N 1024
#define L 3 

#include<stdio.h>
#include<stdlib.h>
#include<time.h>

void initData(float x[N]);
void print(float x[N]);
__global__ void convolve(float *x, float *h, float *y, int fftL);

int floorLog(int num){
	int i=0;
	while(num > 1){
		num /= 2;
		i++;
	}

	return i;
}


int main(){
	float x[N];
	float h[L] = {0,1,1};
	float y[N+L-1] = {0};

	float *dev_x, *dev_h, *dev_y;
	
	hipMalloc((void **)&dev_x, N*sizeof(float));
	hipMalloc((void **)&dev_h, L*sizeof(float));
	hipMalloc((void **)&dev_y, (N+L-1)*sizeof(float));

	initData(x);

	int fftL;
	fftL = ( 1 << (floorLog(L) + 1) );	

	int ovlapSize = L-1;
	int noOfBlocks = (N+L-1+fftL-1)/(fftL-ovlapSize);

	hipMemcpy(dev_x,&x,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_h,&h,L*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,&y,(N+L-1)*sizeof(float),hipMemcpyHostToDevice);
	

	dim3 dimGrid(noOfBlocks,1);
	dim3 dimBlock(fftL,1);

	convolve<<<dimGrid,dimBlock>>>(dev_x dev_h, dev_y,fftL);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	
	cudaMemcoy(&y,dev_y,(N+L-1)*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_h);
	
	print(y);
	
	return 0;
}

void initData(float x[N]){
	int i;
	for(i=0;i<N;i++){
		x[i] = i;
	}
}

void print(float x[N]){
	int i;
	for(i=0;i<N;i++){
		printf("%.3f\t",x[i]);
	}

}

__global__ void convolve(float *x, float *h, float *y, int fftL){
	
	float zph[fftL] = {0};
	float zpx[N+L-1+fftL-1] = {0}	
	int i;
	for(i=0;i<L;i++)
		zph[i] = h[i];

	absId = blockIdx.x*blockDim.x+threadIdx.x;

	if(absId < N) zpx[L-1 + absId] = x[absId];
	
	
	__shared__ float xlocal[fftL] = {0};

}
