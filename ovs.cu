#include "hip/hip_runtime.h"
#define N 1024
#define L 3 

#include<stdio.h>
#include<stdlib.h>
#include<time.h>

void initData(float x[N]);
void print(float x[N]);
__global__ void convolve(float *x, float *h, float *y, int fftL);

int floorLog(int num){
	int i;
	while(num > 1){
		num /= 2;
		i++;
	}

	return i;

}
int main(){
	float x[N];
	float h[L] = {0,1,1};
	float y[N+L-1] = {0};

	float *dev_x, *dev_h, *dev_y;
	
	hipMalloc((void **)&dev_x, N*sizeof(float));
	hipMalloc((void **)&dev_h, L*sizeof(float));
	hipMalloc((void **)&dev_y, (N+L-1)*sizeof(float));

	initData(x);

	hipMemcpy(dev_x,&x,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_h,&h,L*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_y,&y,(N+L-1)*sizeof(float),hipMemcpyHostToDevice);
	
	int fftL;
	

	/*
		decide fftL

   */

	dim3 dimGrid(N/fftL,1);
	dim3 dimBlock(fftL,1);

	convolve<<<dimGrid,dimBlock>>>(dev_x dev_h, dev_y,fftL);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		printf("Error: %s\n", hipGetErrorString(err));
	
	cudaMemcoy(&y,dev_y,(N+L-1)*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_h);
	
	print(y);
	
	return 0;
}

void initData(float x[N]){
	int i;
	for(i=0;i<N;i++){
		x[i] = i;
	}
}

void print(float x[N]){
	int i;
	for(i=0;i<N;i++){
		printf("%.3f\t",x[i]);
	}

}

__global__ void convolve(float *x, float *h, float *y, int fftL){



}
